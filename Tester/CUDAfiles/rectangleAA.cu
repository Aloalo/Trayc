#include "hip/hip_runtime.h"
#include <Trayc/CUDAfiles/helper.h>
#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float3, plane_normal, , );
rtDeclareVariable(float3, recmin, , );
rtDeclareVariable(float3, recmax, , );
 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
	float t = dot(recmin - ray.origin, plane_normal) / dot(ray.direction, plane_normal);
	if(t > ray.tmin && t < ray.tmax)
	{
		float3 p = ray.origin + ray.direction * t;

		if(isBetween(recmin, recmax, p))
			if(rtPotentialIntersection(t))
			{
				shading_normal = geometric_normal = plane_normal;
				rtReportIntersection(0);
			}
	}
}

RT_PROGRAM void bounds(int, float result[6])
{
	optix::Aabb* aabb = (optix::Aabb*)result;
	aabb->set(recmin, recmax);
}

