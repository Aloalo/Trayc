#include "hip/hip_runtime.h"
#include <Trayc/CUDAfiles/phong.h>

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

//
//ADS phong shader with shadows and reflections, no textures
//
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, reflectivity, , );

RT_PROGRAM void closest_hit_phong()
{
    const float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    const float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    const float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
    const float3 hit_point = ray.origin + t_hit * ray.direction;

    const uint2 screen = output_buffer.size();
    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame);

    phongShade(hit_point, Ka, Kd, Ks, ffnormal, phong_exp, seed);
    phongReflect(hit_point, ffnormal, reflectivity);
    prd_radiance.result *= ambientOcclusion(hit_point, ffnormal, seed);
}

RT_PROGRAM void any_hit()
{
    phongShadowed();
}
