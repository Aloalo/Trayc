#include "hip/hip_runtime.h"
/*
* Copyright (c) 2014 Jure Ratkovic
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, boxmin, , );
rtDeclareVariable(float3, boxmax, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

static __device__ float3 boxnormal(float t)
{
    const float3 t0 = (boxmin - ray.origin) / ray.direction;
    const float3 t1 = (boxmax - ray.origin) / ray.direction;
    const float3 neg = make_float3(t == t0.x ? 1:0, t == t0.y ? 1:0, t == t0.z ? 1:0);
    const float3 pos = make_float3(t == t1.x ? 1:0, t == t1.y ? 1:0, t == t1.z ? 1:0);
    return pos - neg;
}

RT_PROGRAM void intersect(int)
{
    const float3 t0 = (boxmin - ray.origin) / ray.direction;
    const float3 t1 = (boxmax - ray.origin) / ray.direction;
    const float3 near = fminf(t0, t1);
    const float3 far = fmaxf(t0, t1);
    const float tmin = fmaxf(near);
    const float tmax = fminf(far);

    if(tmin <= tmax)
    {
        bool check_second = true;
        if(rtPotentialIntersection(tmin))
        {
            shading_normal = geometric_normal = boxnormal(tmin);
            if(rtReportIntersection(0))
                check_second = false;
        } 
        if(check_second)
        {
            if(rtPotentialIntersection(tmax))
            {
                shading_normal = geometric_normal = boxnormal(tmax);
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void bounds(int, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->set(boxmin, boxmax);
}
