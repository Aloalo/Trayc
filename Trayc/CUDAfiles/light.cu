#include "hip/hip_runtime.h"
/*
* Copyright (c) 2014 Jure Ratkovic
*/

#include <Trayc/CUDAfiles/common.h>

using namespace optix;

rtDeclareVariable(int, light_idx, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

RT_PROGRAM void intersect(int)
{
    const float3 center = lights[light_idx].pos;
    const float3 O = ray.origin - center;
    const float3 D = ray.direction;
    const float radius = lights[light_idx].radius;

    const float b = dot(O, D);
    const float c = dot(O, O) - radius * radius;
    const float disc = b * b - c;
    if(disc > 0.0f)
    {
        const float sdisc = sqrtf(disc);
        const float root1 = (-b - sdisc);

        bool check_second = true;
        if(rtPotentialIntersection(root1))
        {
            shading_normal = geometric_normal = (O + root1 * D) / radius;
            if(rtReportIntersection(0))
                check_second = false;
        } 
        if(check_second)
        {
            const float root2 = (-b + sdisc);
            if(rtPotentialIntersection(root2))
            {
                shading_normal = geometric_normal = (O + root2*D)/radius;
                rtReportIntersection(0);
            }
        }
    }
}

RT_PROGRAM void bounds(int, float result[6])
{
    const float3 cen = lights[light_idx].pos;
    const float3 rad = make_float3(lights[light_idx].radius);

    optix::Aabb* aabb = (optix::Aabb*)result;

    if(rad.x > 0.0f  && !isinf(rad.x))
    {
        aabb->m_min = cen - rad;
        aabb->m_max = cen + rad;
    }
    else
    {
        aabb->invalidate();
    }
}
