#include "hip/hip_runtime.h"
/*
* Copyright (c) 2014 Jure Ratkovic
*/

#include <Trayc/CUDAfiles/common.h>

using namespace optix;

rtDeclareVariable(float, bloom_exponent, , );

rtDeclareVariable(int, light_idx, , );
rtDeclareVariable(float, light_brightness, attribute light_brightness, );
rtDeclareVariable(float, importance_cutoff, , );

RT_PROGRAM void intersect(int)
{

    const float3 center = lights[light_idx].pos;
    const float radius = lights[light_idx].radius;
    const float bloomFactor = 2.0f;
    const float bloomRadius = bloomFactor * radius;

    const float t = dot(ray.direction, center - ray.origin);
    const float3 planePoint = ray.origin + t * ray.direction;

    const float hitDistToCenter2 = dot(planePoint - center, planePoint - center);
    const float bloomRadius2 = bloomRadius * bloomRadius;
    const float adjusted_t = t + sqrtf(bloomRadius2 - hitDistToCenter2);

    
    light_brightness = 1.0f;
    if(hitDistToCenter2 <= radius * radius)
    {
        if(rtPotentialIntersection(adjusted_t))
            rtReportIntersection(0);
    }
    else if(hitDistToCenter2 <= bloomRadius2 || fmaxf(lights[light_idx].color) > importance_cutoff)
    {
        if(rtPotentialIntersection(adjusted_t))
        {
            const float bloom = 1.0f - (sqrtf(hitDistToCenter2) - radius) / (radius * (bloomFactor - 1));
            light_brightness = powf(bloom, bloom_exponent);
            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void bounds(int, float result[6])
{
    const float3 cen = lights[light_idx].pos;
    const float3 rad = make_float3(lights[light_idx].radius);

    optix::Aabb* aabb = (optix::Aabb*)result;

    if(rad.x > 0.0f  && !isinf(rad.x))
    {
        aabb->m_min = cen - rad;
        aabb->m_max = cen + rad;
    }
    else
    {
        aabb->invalidate();
    }
}
