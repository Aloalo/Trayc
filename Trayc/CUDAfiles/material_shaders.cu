#include "hip/hip_runtime.h"
#include "phong.h"

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

//
// Transparent object shadows, no textures
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void any_hit_glass()
{
    const float3 world_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    const float nDi = fabs(dot(world_normal, ray.direction));

    prd_shadow.attenuation *= 1.0f - optix::fresnel_schlick(nDi, 5.0f, 1.0f - shadow_attenuation, make_float3(1.0f)).x;

    rtIgnoreIntersection();
}

//
// Glass shader, no textures
//
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float, fresnel_exponent, , );
rtDeclareVariable(float, fresnel_minimum, , );
rtDeclareVariable(float, fresnel_maximum, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(float3, refraction_color, , );
rtDeclareVariable(float3, reflection_color, , );
rtDeclareVariable(float3, extinction_constant, , );

RT_PROGRAM void closest_hit_glass()
{
    const float3 h = ray.origin + t_hit * ray.direction;
    const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
    const float3 &i = ray.direction;// incident direction

    float reflection = 1.0f;
    float3 result = make_float3(0.0f);

    const float3 beer_attenuation = dot(n, ray.direction) > 0.0f ? 
                                        exp(extinction_constant * t_hit) :
                                        make_float3(1.0f);

    if(prd_radiance.depth < max_depth)
    {
        float3 t;
        if(refract(t, i, n, refraction_index))
        {
            // check for external or internal reflection
            float cos_theta = dot(i, n);
            if(cos_theta < 0.0f)
                cos_theta = -cos_theta;
            else
                cos_theta = dot(t, n);

            reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

            const float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
            if(importance > importance_cutoff)
            {
                optix::Ray ray(h, t, radiance_ray_type, scene_epsilon);
                PerRayData_radiance refr_prd;
                refr_prd.depth = prd_radiance.depth+1;
                refr_prd.importance = importance;

                rtTrace(top_object, ray, refr_prd);
                result += (1.0f - reflection) * refraction_color * refr_prd.result;
            }
            else
                result += (1.0f - reflection) * refraction_color * cutoff_color;
        }

        const float3 r = reflect(i, n);

        const float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
        if(importance > importance_cutoff)
        {
            const optix::Ray ray(h, r, radiance_ray_type, scene_epsilon);
            PerRayData_radiance refl_prd;
            refl_prd.depth = prd_radiance.depth + 1;
            refl_prd.importance = importance;

            rtTrace(top_object, ray, refl_prd);
            result += reflection * reflection_color * refl_prd.result;
        }
        else
            result += reflection * reflection_color * cutoff_color;
    }

    result = result * beer_attenuation;
    prd_radiance.result = result;
}

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, reflectivity, , );     
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> diffuse_map;
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> specular_map;

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 

//
//solid mesh with textures and reflectivity
//
RT_PROGRAM void closest_hit_mesh()
{
    const float4 pKd = tex2D(diffuse_map, texcoord.x, texcoord.y);
    if(prd_radiance.depth < max_depth && pKd.w < importance_cutoff)
    {
        const optix::Ray newray(ray.origin + t_hit * ray.direction, ray.direction, radiance_ray_type, scene_epsilon);
        prd_radiance.depth++;
        rtTrace(top_object, newray, prd_radiance);

        return;
    }
    
    const float3 world_shading_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    const float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
    const float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

    const float4 pKs = tex2D(specular_map, texcoord.x, texcoord.y);
    
    //phongShade(ffnormal, make_float3(0.0f), make_float3(0.0f), make_float3(0.0f), phong_exp, reflectivity);
    //phongShade(make_float3(abs(ffnormal.x), abs(ffnormal.y), abs(ffnormal.z)), make_float3(0.0f), make_float3(0.0f), make_float3(0.0f), phong_exp, reflectivity);
    phongShade(make_float3(pKd) * Ka, make_float3(pKd) * Kd, make_float3(pKs) * Ks, ffnormal, pKs.w * 255.0f, reflectivity);
}

//
// Terminates and fully attenuates ray after any hit
//
RT_PROGRAM void any_hit_solid()
{
    const float opacity = tex2D(diffuse_map, texcoord.x, texcoord.y).w;
    if(opacity < importance_cutoff)
        rtIgnoreIntersection();
    phongShadowed();
}
