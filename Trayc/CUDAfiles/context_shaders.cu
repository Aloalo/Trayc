#include "hip/hip_runtime.h"
#include "phong.h"

//
// Pinhole/DOF camera implementation
//
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

rtDeclareVariable(int, renderingDivisionLevel, , "Number of horizontal stripes");
rtDeclareVariable(int, myStripe, , "Current stripe");

rtBuffer<uchar4, 2> output_buffer;

rtDeclareVariable(int, AAlevel, , );
rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float, focal_length, , );
rtDeclareVariable(int, dof_samples, , );

RT_PROGRAM void dof_camera()
{
    const float2 screen = make_float2(output_buffer.size() * AAlevel);
    float3 result = make_float3(0.0f);
    const uint2 newLaunchIndex = make_uint2(launch_index.x, launch_index.y + myStripe * output_buffer.size().y / renderingDivisionLevel);
    unsigned int seed = (launch_index.x * 1920 + launch_index.y) * launch_index.x * launch_index.y;
    float count = 0.0f;

    for(int i = 0; i < AAlevel; ++i)
        for(int j = 0; j < AAlevel; ++j)
        {
            const float2 d = make_float2(AAlevel * newLaunchIndex.x + i, AAlevel * newLaunchIndex.y + j) / screen * 2.f - 1.f;
            const float3 ray_direction = normalize(d.x * U + d.y * V + W);

            const optix::Ray ray(eye, ray_direction, radiance_ray_type, scene_epsilon);

            PerRayData_radiance prd;
            prd.importance = 1.f;
            prd.depth = 0;

            rtTrace(top_object, ray, prd);
            result += prd.result;
            count += 1.0f;

            for(int k = 1; k < dof_samples; ++k)
            {
                const float2 d_dof = make_float2(AAlevel * newLaunchIndex.x + i, AAlevel * newLaunchIndex.y + j) / screen * 2.f - 1.f;
                const float3 ray_direction_dof = normalize(d_dof.x * U + d_dof.y * V + W);

                optix::Ray ray_dof(eye, ray_direction_dof, radiance_ray_type, scene_epsilon);

                const float2 circPoint = make_float2(rnd(seed) * 2.0f - 1.0f, rnd(seed) * 2.0f - 1.0f);
                const float3 apertureOffset = make_float3(circPoint.x * aperture_radius, circPoint.y * aperture_radius, 0.0f);
                ray_dof.origin += apertureOffset;
                ray_dof.direction *= focal_length;
                ray_dof.direction -= apertureOffset;
                ray_dof.direction = normalize(ray_dof.direction);

                PerRayData_radiance prd;
                prd.importance = 1.f;
                prd.depth = 0;

                rtTrace(top_object, ray_dof, prd);
                count += 1.0f;
                result += prd.result;
            }
        }

    output_buffer[newLaunchIndex] = make_color(result / count);
}

//
// Enviormement map
//
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> envmap;
RT_PROGRAM void envmap_miss()
{
    const float theta = atan2f(ray.direction.x, ray.direction.z);
    const float phi = M_PIf * 0.5f - acosf(ray.direction.y);
    const float u = (theta + M_PIf) * (0.5f * M_1_PIf);
    const float v = 0.5f * (1.0f + sinf(phi));
    prd_radiance.result = make_float3(tex2D(envmap, u, v));
}

//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, miss_color, , );
RT_PROGRAM void miss()
{
    prd_radiance.result = miss_color;
}

//
// Returns color from [miss_min, miss_max] lineary interpolated across ray inclination
//
rtDeclareVariable(float3, miss_min, , );
rtDeclareVariable(float3, miss_max, , );
RT_PROGRAM void gradient_miss()
{
    const float phi = asinf(ray.direction.y);
    prd_radiance.result = 2.0f * phi / M_PIf * (miss_max - miss_min) + miss_min;
}

//
// Set pixel to solid color upon failure
//
rtDeclareVariable(float3, bad_color, , );

RT_PROGRAM void exception()
{
    const uint2 newLaunchIndex = make_uint2(launch_index.x, launch_index.y + myStripe * output_buffer.size().y / renderingDivisionLevel);
    output_buffer[newLaunchIndex] = make_color(bad_color);
}
