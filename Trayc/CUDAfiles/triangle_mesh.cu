#include "hip/hip_runtime.h"
/*
* Copyright (c) 2014 Jure Ratkovic
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float3> vertex_buffer;     
rtBuffer<float3> normal_buffer;
rtBuffer<float3> tangent_buffer;
rtBuffer<float3> bitangent_buffer;
rtBuffer<float2> texcoord_buffer;
rtBuffer<int3> index_buffer;
rtTextureSampler<uchar4, 2, hipReadModeNormalizedFloat> normal_map;

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void intersect(int primIdx)
{
    const int3 &idx = index_buffer[primIdx];

    const float3 &p0 = vertex_buffer[idx.x];
    const float3 &p1 = vertex_buffer[idx.y];
    const float3 &p2 = vertex_buffer[idx.z];

    float3 n;
    float t, beta, gamma;
    if(intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma))
    {
        if(rtPotentialIntersection(t))
        {
            if(texcoord_buffer.size() == 0)
                texcoord = make_float3(0.0f, 0.0f, 0.0f);
            else 
            {
                const float2 &t0 = texcoord_buffer[idx.x];
                const float2 &t1 = texcoord_buffer[idx.y];
                const float2 &t2 = texcoord_buffer[idx.z];
                texcoord = make_float3(t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma));
            }
            
            geometric_normal = normalize(n);

            if(normal_buffer.size() == 0)
                shading_normal = geometric_normal;
            else
            {
                shading_normal = normalize(normal_buffer[idx.y] * beta + 
                    normal_buffer[idx.z] * gamma + normal_buffer[idx.x] * (1.0f - beta - gamma));
        
                if(tangent_buffer.size() > 0)
                {
                    const float3 shading_tangent = normalize(tangent_buffer[idx.y] * beta +
                        tangent_buffer[idx.z] * gamma + tangent_buffer[idx.x] * (1.0f - beta - gamma));
                    const float3 shading_bitangent = normalize(bitangent_buffer[idx.y] * beta +
                        bitangent_buffer[idx.z] * gamma + bitangent_buffer[idx.x] * (1.0f - beta - gamma));
                    Matrix3x3 tbni;
                    tbni.setCol(0, shading_tangent);
                    tbni.setCol(1, shading_bitangent);
                    tbni.setCol(2, shading_normal);
                    shading_normal = tbni * normalize(make_float3(tex2D(normal_map, texcoord.x, texcoord.y)) * 2.f - 1.f);
                }
            }

            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void bounds(int primIdx, float result[6])
{  
    const int3 &idx = index_buffer[primIdx];

    const float3 &v0 = vertex_buffer[idx.x];
    const float3 &v1 = vertex_buffer[idx.y];
    const float3 &v2 = vertex_buffer[idx.z];
    const float area = length(cross(v1 - v0, v2 - v0));

    optix::Aabb *aabb = (optix::Aabb*)result;

    if(area > 0.0f && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    }
    else
        aabb->invalidate();
}

