#include "hip/hip_runtime.h"
/*
* Copyright (c) 2014 Jure Ratkovic
*/

#include <optix_world.h>

using namespace optix;

rtDeclareVariable(float4, sphere, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect(int)
{
    const float3 center = make_float3(sphere);
    const float3 O = ray.origin - center;
    const float3 D = ray.direction;
    const float radius = sphere.w;

    const float b = dot(O, D);
    const float c = dot(O, O) - radius * radius;
    const float disc = b*b-c;
    if(disc > 0.0f)
    {
        const float sdisc = sqrtf(disc);
        const float root1 = (-b - sdisc);

        bool check_second = true;
        if(rtPotentialIntersection(root1))
        {
            shading_normal = geometric_normal = (O + root1 * D) / radius;
            if(rtReportIntersection(0))
                check_second = false;
        } 
        if(check_second)
        {
            const float root2 = (-b + sdisc);
            if(rtPotentialIntersection(root2))
            {
                shading_normal = geometric_normal = (O + root2*D)/radius;
                rtReportIntersection(0);
            }
        }
    }
}


RT_PROGRAM void bounds(int, float result[6])
{
    const float3 cen = make_float3( sphere );
    const float3 rad = make_float3( sphere.w );

    optix::Aabb* aabb = (optix::Aabb*)result;

    if(rad.x > 0.0f  && !isinf(rad.x))
    {
        aabb->m_min = cen - rad;
        aabb->m_max = cen + rad;
    }
    else
    {
        aabb->invalidate();
    }
}

